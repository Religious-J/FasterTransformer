#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <assert.h>

#include "src/fastertransformer/kernels/beam_search_penalty_kernels.h"
#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"

namespace fastertransformer {

template<typename T>
__global__ void add_bias_temperature(T*          logits,
                                     const T*    bias,
                                     const int   batch_size,
                                     const int   beam_width,
                                     const int   vocab_size,
                                     const int   vocab_size_padded,
                                     const float temperature)
{
    int tid  = threadIdx.x;
    int bid  = blockIdx.x;
    int bbid = blockIdx.y;

    logits += bbid * vocab_size_padded;

    const T MASK_VAL = (std::is_same<T, half>::value) ? -HALF_FLT_MAX : -FLT_MAX;
    const T inv_temp = static_cast<T>(1.0f / (temperature + 1e-6f));
    for (int i = tid + bid * blockDim.x; i < vocab_size_padded; i += blockDim.x * gridDim.x) {
        if (i < vocab_size) {
            T bias_val = bias == nullptr ? (T)(0.0f) : bias[i];
            logits[i]  = (logits[i] + bias_val) * inv_temp;
        }
        else {
            logits[i] = MASK_VAL;
        }
    }
}

template<>
__global__ void add_bias_temperature(half2*       logits,
                                     const half2* bias,
                                     const int    batch_size,
                                     const int    beam_width,
                                     const int    vocab_size,
                                     const int    vocab_size_padded,
                                     const float  temperature)
{
    assert(vocab_size % 2 == 0);
    assert(vocab_size_padded % 2 == 0);

    const int tid  = threadIdx.x;
    const int bid  = blockIdx.x;
    const int bbid = blockIdx.y;

    const half2 mask_val = __float2half2_rn(-HALF_FLT_MAX);
    const half2 inv_temp = __float2half2_rn(1.0f / (temperature + 1e-6f));

    const int half_vocab_size        = vocab_size / 2;
    const int half_vocab_size_padded = vocab_size_padded / 2;

    logits += bbid * half_vocab_size_padded;
    for (int index = tid + bid * blockDim.x; index < half_vocab_size_padded; index += blockDim.x * gridDim.x) {
        int   vocab_idx = index % half_vocab_size_padded;
        half2 logit     = vocab_idx < half_vocab_size ? __ldg(&logits[index]) : mask_val;
        if (vocab_idx < half_vocab_size) {
            if (bias != nullptr) {
                logit = __hadd2(logit, bias[vocab_idx]);
            }
            logit = __hmul2(logit, inv_temp);
        }
        logits[index] = logit;
    }
}

/*
    float logits[10] = {0.2, 0.8, 0.3, 0.1, 0.5, 0.9, 0.4, 0.7, 0.6, 0.05};  // logits 数组
    int current_ids[1] = {2};                                                // 当前步骤生成的 token ID
    int previous_ids[3] = {1, 3, 4};                                         // 之前步骤的 token ID
    int input_lengths[1] = {3};                                              // 输入长度
    int step = 3;                                                            // 当前生成步骤
    int max_input_length = 5;                                                // 最大输入长度
    float repetition_penalty = 1.5;                                          // 惩罚系数
    int is_additive = 1;                                                     // 使用加法惩罚
*/

// 对之前所有的词做惩罚 ？？ 对之前出现的所有词 -？
// C 函数版本的 apply_repetition_penalty + batch_size=1 beam_width=1
void apply_repetition_penalty(float* logits,
                              int vocab_size_padded,
                              int step,                     // 当前生成步骤
                              const int* current_ids,       //  当前步骤生成的 token ID
                              const int* previous_ids,      // 之前步骤的 token ID
                              const int* input_lengths,     // 输入长度
                              int max_input_length,         // 最大输入长度
                              float repetition_penalty,     // 惩罚系数
                              int is_additive               // 使用加法惩罚?
                              )
{
    assert(step > 0);

    // 当前 ID 和长度
    int prev_id = current_ids[0];
    int input_length = (input_lengths != NULL) ? input_lengths[0] : max_input_length;

    // 创建局部数组用于存储调整后的 logits 和对应的索引
    float penalty_logits[step];
    int penalty_indices[step];   // 惩罚指标

    // 初始化
    penalty_indices[step - 1] = prev_id;
    float prev_logit = logits[prev_id];
    if (is_additive) {
        penalty_logits[step - 1] = prev_logit - repetition_penalty;
    } else {
        penalty_logits[step - 1] = (prev_logit > 0) ? prev_logit / repetition_penalty : prev_logit * repetition_penalty;
    }

    // 处理历史步骤
    if (step > 1) {
        for (int i = step - 2; i >= 0; --i) {
            // 跳过填充的 token
            if (i >= input_length && i < max_input_length) {
                continue;
            }

            // 获取之前的 ID 和 logit
            prev_id = previous_ids[i];
            prev_logit = logits[prev_id];

            penalty_indices[i] = prev_id;    // 用这样记录，感觉最后用 previous_ids 反向遍历感觉也可以 
            if (is_additive) {
                penalty_logits[i] = prev_logit - repetition_penalty;
            } else {
                penalty_logits[i] = (prev_logit > 0) ? prev_logit / repetition_penalty : prev_logit * repetition_penalty;
            }
        }
    }

    // 写回到 logits
    for (int i = 0; i < step; i++) {
        if (i >= input_length && i < max_input_length) {
            continue;
        }
        logits[penalty_indices[i]] = penalty_logits[i];
    }
}


// CPU
void apply_repetition_penalty_CPU(float* logits,
                              const int batch_size,
                              const int beam_width,
                              const int vocab_size,
                              const int vocab_size_padded,
                              const int step,
                              const int* current_ids,
                              const int* previous_ids,
                              const int* parent_ids,
                              const int* input_lengths,
                              const int max_input_length,
                              const float repetition_penalty) {
    assert(step > 0);

    const int bbsize = batch_size * beam_width;

    for (int bbid = 0; bbid < bbsize; ++bbid) {
        int input_length = (input_lengths != NULL) ? input_lengths[bbid] : max_input_length;
        float penalty_logits[step];
        int penalty_indices[step];

        float repet_penalty = repetition_penalty;
        int prev_id = current_ids[bbid];
        float prev_logit = logits[prev_id];
        penalty_indices[step - 1] = prev_id;

        // Apply penalty for the current token
        if (prev_logit > 0) {
            penalty_logits[step - 1] = IS_ADDITIVE ? (prev_logit - repet_penalty) : (prev_logit / repet_penalty);
        } else {
            penalty_logits[step - 1] = IS_ADDITIVE ? (prev_logit - repet_penalty) : (prev_logit * repet_penalty);
        }

        // Process previous steps
        if (step > 1) {
            int parent_beam = bbid % beam_width;
            for (int i = step - 2; i >= 0; --i) {
                if (i >= input_length && i < max_input_length) {
                    continue;
                }
                parent_beam = parent_ids[i * bbsize + bbid];
                prev_id = previous_ids[i * bbsize + bbid];
                prev_logit = logits[prev_id];
                penalty_indices[i] = prev_id;

                if (prev_logit > 0) {
                    penalty_logits[i] = IS_ADDITIVE ? (prev_logit - repet_penalty) : (prev_logit / repet_penalty);
                } else {
                    penalty_logits[i] = IS_ADDITIVE ? (prev_logit - repet_penalty) : (prev_logit * repet_penalty);
                }
            }
        }

        // Update logits
        for (int i = 0; i < step; ++i) {
            if (i >= input_length && i < max_input_length) {
                continue;
            }
            logits[penalty_indices[i]] = penalty_logits[i];
        }
    }
}

template<typename T, bool IS_ADDITIVE>
__global__ void apply_repetition_penalty(T*          logits,
                                         const int   batch_size,
                                         const int   beam_width,
                                         const int   vocab_size,
                                         const int   vocab_size_padded,
                                         const int   step,
                                         const int*  current_ids,
                                         const int*  previous_ids,
                                         const int*  parent_ids,
                                         const int*  input_lengths,
                                         const int   max_input_length,
                                         const float repetition_penalty)
{
    assert(step > 0);

    const int tid      = threadIdx.x;
    const int bbid     = blockIdx.x;
    const int batch_id = bbid / beam_width;
    const int bbsize   = batch_size * beam_width;

    logits += bbid * vocab_size_padded;
    extern __shared__ char sbuf[];
    T*                     penalty_logits = reinterpret_cast<T*>(sbuf);
    // prevent misaligment when sizeof(T) = 2
    int*      penalty_indices = reinterpret_cast<int*>(sbuf + (sizeof(T) * step + 31) / 32 * 32);
    const int input_length    = (input_lengths != nullptr) ? input_lengths[bbid] : max_input_length;
    if (tid == 0) {   // only tid 0
        T   repet_penalty         = static_cast<T>(repetition_penalty);
        int prev_id               = current_ids[bbid];
        T   prev_logit            = logits[prev_id];
        penalty_indices[step - 1] = prev_id;

        if (IS_ADDITIVE) {
            penalty_logits[step - 1] = prev_logit - repet_penalty;
        }
        else {
            penalty_logits[step - 1] = prev_logit > T(0) ? prev_logit / repet_penalty : prev_logit * repet_penalty;
        }
        if (step > 1) {
            int parent_beam = bbid % beam_width;
            for (int i = step - 2; i >= 0; --i) {
                // Skip the padded tokens.
                if (i >= input_length && i < max_input_length) {
                    continue;
                }
                parent_beam        = parent_ids[i * bbsize + batch_id * beam_width + parent_beam];
                prev_id            = previous_ids[i * bbsize + batch_id * beam_width + parent_beam];
                prev_logit         = logits[prev_id];
                penalty_indices[i] = prev_id;
                if (IS_ADDITIVE) {
                    penalty_logits[i] = prev_logit - repet_penalty;
                }
                else {
                    penalty_logits[i] = prev_logit > T(0) ? prev_logit / repet_penalty : prev_logit * repet_penalty;
                }
            }
        }
    }
    __syncthreads();
    // 就赋值并行处理一下？ YES
    for (int i = tid; i < step; i += blockDim.x) {
        if (i >= input_length && i < max_input_length) {
            continue;
        }
        logits[penalty_indices[i]] = penalty_logits[i];
    }
}

template<typename T>
__global__ void apply_min_length_penalty(T*         logits,
                                         const int  min_length,
                                         const int* end_ids,
                                         const int* sequence_lengths,
                                         const int  max_input_length,
                                         const int  beam_width,
                                         const int  vocab_size_padded)
{
    int bbid = threadIdx.x + blockIdx.x * blockDim.x;  // batch-beam index
    int bid  = bbid / beam_width;                      // batch index
    // We need +1 because sequence_lengths = max_input_length + num_gen_tokens - 1,
    // which is equal to the length of k/v caches.
    if (sequence_lengths[bbid] + 1 - max_input_length < min_length) {
        T mask_val                                      = (std::is_same<T, half>::value) ? -HALF_FLT_MAX : -FLT_MAX;
        logits[bbid * vocab_size_padded + end_ids[bid]] = mask_val;
    }
}

template<typename T>
void invokeAddBiasApplyPenalties(int                         step,
                                 T*                          logits,
                                 const int*                  current_ids,
                                 const int*                  previous_ids,
                                 const int*                  parent_ids,
                                 const int*                  input_lengths,
                                 const int*                  sequence_lengths,
                                 const T*                    bias,
                                 const int                   ite,
                                 const int                   max_input_length,
                                 const int                   local_batch_size,
                                 const int                   batch_size,
                                 const int                   beam_width,
                                 const int                   vocab_size,
                                 const int                   vocab_size_padded,
                                 const int*                  end_ids,
                                 const float                 temperature,
                                 const float                 repetition_penalty,
                                 const RepetitionPenaltyType repetition_penalty_type,
                                 const int                   min_length,
                                 hipStream_t                stream)
{
    if (bias != nullptr || temperature != 1.0f || vocab_size != vocab_size_padded) {
        dim3 block(512);
        if (std::is_same<T, half>::value && vocab_size % 2 == 0 && vocab_size_padded % 2 == 0) {
            dim3 grid((vocab_size_padded / 2 + block.x - 1) / block.x, beam_width * local_batch_size);
            // add_bias_temperature：负责将偏置和温度应用到 logits
            add_bias_temperature<<<grid, block, 0, stream>>>(reinterpret_cast<half2*>(logits),
                                                             reinterpret_cast<const half2*>(bias),
                                                             batch_size,
                                                             beam_width,
                                                             vocab_size,
                                                             vocab_size_padded,
                                                             temperature);
        }
        else {
            dim3 grid((vocab_size_padded + block.x - 1) / block.x, beam_width * local_batch_size);
            add_bias_temperature<<<grid, block, 0, stream>>>(
                logits, bias, batch_size, beam_width, vocab_size, vocab_size_padded, temperature);
        }
    }

    if (repetition_penalty_type != RepetitionPenaltyType::None && step > 0) {
        if (repetition_penalty != getDefaultPenaltyValue(repetition_penalty_type)) {
            size_t smem_size = (sizeof(T) * step + 31) / 32 * 32 + sizeof(int) * step;
            dim3   block(256);
            dim3   grid(beam_width * local_batch_size);
            // apply_repetition_penalty：负责对 logits 应用重复惩罚，依据不同的惩罚方式（乘法或加法）进行调整
            if (repetition_penalty_type == RepetitionPenaltyType::Multiplicative) {
                apply_repetition_penalty<T, false>
                    <<<grid, block, smem_size, stream>>>(logits,
                                                         batch_size,
                                                         beam_width,
                                                         vocab_size,
                                                         vocab_size_padded,
                                                         step,
                                                         current_ids,
                                                         previous_ids,
                                                         // TODO(jaedeokk):
                                                         //   Remove (+ite ...) by getting parent_ids with offset
                                                         //   and then remove 'ite' argument from the function.
                                                         parent_ids + ite * beam_width * local_batch_size,
                                                         input_lengths,
                                                         max_input_length,
                                                         repetition_penalty);
            }
            else if (repetition_penalty_type == RepetitionPenaltyType::Additive) {
                apply_repetition_penalty<T, true>
                    <<<grid, block, smem_size, stream>>>(logits,
                                                         batch_size,
                                                         beam_width,
                                                         vocab_size,
                                                         vocab_size_padded,
                                                         step,
                                                         current_ids,
                                                         previous_ids,
                                                         parent_ids + ite * beam_width * local_batch_size,
                                                         input_lengths,
                                                         max_input_length,
                                                         repetition_penalty);
            }
        }
    }

    if (step - max_input_length < min_length) {
        FT_CHECK_WITH_INFO(sequence_lengths != nullptr, "Need sequence_lengths to apply min length penlaty");
        FT_CHECK_WITH_INFO(end_ids != nullptr, "Need end_id to apply min length penlaty");

        const int block_size = min(local_batch_size * beam_width, 1024);
        const int grid_size  = (local_batch_size * beam_width + block_size - 1) / block_size;
        // apply_min_length_penalty：确保生成的序列满足最小长度要求
        apply_min_length_penalty<<<grid_size, block_size, 0, stream>>>(
            logits, min_length, end_ids, sequence_lengths, max_input_length, beam_width, vocab_size_padded);
    }
}

template void invokeAddBiasApplyPenalties(int                         step,
                                          float*                      logits,
                                          const int*                  current_ids,
                                          const int*                  previous_ids,
                                          const int*                  parent_ids,
                                          const int*                  input_lengths,
                                          const int*                  sequence_lengths,
                                          const float*                bias,
                                          const int                   ite,
                                          const int                   max_input_length,
                                          const int                   local_batch_size,
                                          const int                   batch_size,
                                          const int                   beam_width,
                                          const int                   vocab_size,
                                          const int                   vocab_size_padded,
                                          const int*                  end_ids,
                                          const float                 temperature,
                                          const float                 repetition_penalty,
                                          const RepetitionPenaltyType repetition_penalty_type,
                                          const int                   min_length,
                                          hipStream_t                stream);

template void invokeAddBiasApplyPenalties(int                         step,
                                          half*                       logits,
                                          const int*                  current_ids,
                                          const int*                  previous_ids,
                                          const int*                  parent_ids,
                                          const int*                  input_lengths,
                                          const int*                  sequence_lengths,
                                          const half*                 bias,
                                          const int                   ite,
                                          const int                   max_input_length,
                                          const int                   local_batch_size,
                                          const int                   batch_size,
                                          const int                   beam_width,
                                          const int                   vocab_size,
                                          const int                   vocab_size_padded,
                                          const int*                  end_ids,
                                          const float                 temperature,
                                          const float                 repetition_penalty,
                                          const RepetitionPenaltyType repetition_penalty_type,
                                          const int                   min_length,
                                          hipStream_t                stream);

}  // namespace fastertransformer
