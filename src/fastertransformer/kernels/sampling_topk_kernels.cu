#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdexcept>
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "src/fastertransformer/kernels/reduce_kernel_utils.cuh"
#include "src/fastertransformer/kernels/sampling_topk_kernels.h"

namespace fastertransformer {

__global__ void curandInitialize(hiprandState_t* state, const int size, const unsigned long long random_seed)
{
    if (threadIdx.x + blockIdx.x * blockDim.x < size) {
        hiprand_init(random_seed, 0, 0, &state[blockIdx.x * blockDim.x + threadIdx.x]);
    }
}

void invokeCurandInitialize(hiprandState_t*           state,
                            const size_t             batch_size,
                            const unsigned long long random_seed,
                            hipStream_t             stream)
{
    dim3 block(256);
    dim3 grid((int)(ceil(batch_size * 1.0 / 256)));
    curandInitialize<<<grid, block, 0, stream>>>(state, batch_size, random_seed);
}

__global__ void curandBatchInitialize(hiprandState_t* states, const int size, const unsigned long long* random_seeds)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        hiprand_init(random_seeds[idx], 0, 0, &states[idx]);
    }
}

void invokeCurandBatchInitialize(hiprandState_t*            states,
                                 const size_t              batch_size,
                                 const unsigned long long* random_seeds,
                                 hipStream_t              stream)
{
    dim3 block(256);
    dim3 grid((int)(ceil(batch_size * 1.0 / 256)));
    curandBatchInitialize<<<grid, block, 0, stream>>>(states, batch_size, random_seeds);
}

template<typename T>
__global__ void addBiasEndMask(T*          logits,
                               const T*    bias,
                               const int*  end_ids,
                               const bool* finished,
                               const int   vocab_size,
                               const int   vocab_size_padded)
{
    int  bid    = blockIdx.x;
    bool finish = finished != nullptr ? finished[bid] : false;
    int  offset = bid * vocab_size_padded;

    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;
    for (int tid = threadIdx.x; tid < vocab_size_padded; tid += blockDim.x) {
        if (tid >= vocab_size) {
            logits[offset + tid] = -MAX_T_VAL;
        }
        else if (finish) {
            logits[offset + tid] = (tid == end_ids[bid]) ? MAX_T_VAL : -MAX_T_VAL;
        }
        else {
            if (bias != nullptr) {
                logits[offset + tid] += bias[tid];
            }
        }
    }
}

template<typename T>
void invokeAddBiasEndMask(T*           logits,
                          const T*     bias,
                          const int*   end_ids,
                          const bool*  finished,
                          const int    batch_size,
                          const int    vocab_size,
                          const int    vocab_size_padded,
                          hipStream_t stream)
{
    dim3 grid(batch_size);
    dim3 block(min(vocab_size_padded, 1024));
    /*n is the vocab_size, e.g., 30000, 7000.... vocab_size is usually very big. */
    addBiasEndMask<<<grid, block, 0, stream>>>(logits, bias, end_ids, finished, vocab_size, vocab_size_padded);
}

template void invokeAddBiasEndMask(float*       logits,
                                   const float* bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    batch_size,
                                   const int    vocab_size,
                                   const int    vocab_size_padded,
                                   hipStream_t stream);

template void invokeAddBiasEndMask(half*        logits,
                                   const half*  bias,
                                   const int*   end_ids,
                                   const bool*  finished,
                                   const int    batch_size,
                                   const int    vocab_size,
                                   const int    vocab_size_padded,
                                   hipStream_t stream);

/*
PS
    在经典 Softmax 中，需要通过 e^{z_i}/sum(e^{z}) 显式地除以总和 sum(e^{z})，以确保归一化后的值在 [0, 1] 之间。
    而在这个 Top-k 采样代码中，没有显式地除以 s_sum 
    因为在采样时 rand_num 和 s_val2 都是在同一个归一化总和的范围内，整个操作逻辑已经间接地完成了归一化。
    我们通过逐渐减去概率值直到随机数 rand_num 变为负数时停止，便实现了概率分布下的采样。
*/

// 从给定的预测概率（log_probs）中选取前 k 个概率最高的值
// Top-k 搜索：通过 CUB 的 BlockReduce 和 TopK_2，每个 block 对 log_probs 中的词汇表进行并行扫描，找到 top-k 概率最大值及其对应的索引
// 并将选出的概率值设置为负无穷大，以避免重复选择。
template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topk_stage1(const T* __restrict log_probs,
                            T*          tmp_log_probs,
                            int*        topk_tmp_id_buf,
                            T*          topk_tmp_val_buf,
                            const bool* finished,
                            const int   max_top_k,
                            const int*  top_ks,
                            const int   vocab_size,
                            const int*  end_ids,
                            const bool* skip_decode)
{
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage     temp_storage;

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    const int batch_id = bid / BLOCKS_PER_BEAM_;  // row id for log_probs
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }
    // 当前块在当前 beam 中的索引
    const int block_lane = bid % BLOCKS_PER_BEAM_;                              // block id for a beam
    // top-k 的值
    const int k          = (top_ks != nullptr) ? top_ks[batch_id] : max_top_k;  // batch_id = batch index

    const int tmp_log_buf_index  = batch_id * vocab_size;
    const int tmp_topk_buf_index = batch_id * BLOCKS_PER_BEAM_ * max_top_k + block_lane * k;

    TopK_2<T>  partial;
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    // 如果该 batch 已经完成（比如已经生成结束符），直接处理结束符
    if (finished != nullptr && finished[batch_id] == true) {
        if (tid < k) {
            const int index = tmp_topk_buf_index + tid;
            if (block_lane == 0 && tid == 0) {
                const int end_id        = end_ids[batch_id];
                topk_tmp_id_buf[index]  = tmp_log_buf_index + end_id;
                topk_tmp_val_buf[index] = log_probs[tmp_log_buf_index + end_id];
            }
            else {
                topk_tmp_id_buf[index]  = -1;
                topk_tmp_val_buf[index] = -MAX_T_VAL;
            }
        }
        return;
    }
    // 将 log_probs 拷贝到 tmp_log_probs 缓存中
    for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
         elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
        int index            = elem_id + tmp_log_buf_index;
        tmp_log_probs[index] = log_probs[index];
    }

    // 开始选择 top-k
    for (int ite = 0; ite < k; ite++) {
        partial.init();  // 初始化 partial 变量用于存储当前线程的 top-k 值
#pragma unroll
        for (int elem_id = tid + block_lane * BLOCK_SIZE_; elem_id < vocab_size;
             elem_id += BLOCK_SIZE_ * BLOCKS_PER_BEAM_) {
            int index = elem_id + tmp_log_buf_index;
            partial.insert(tmp_log_probs[index], index);
        }

        // 通过 BlockReduce 在 block 内部进行 reduce 操作，找到 top-k 值
        TopK_2<T> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<T>);

        // 线程 0 记录 top-k 结果
        if (tid == 0) {
            const int index         = tmp_topk_buf_index + ite;
            topk_tmp_id_buf[index]  = total.p;      // 保存当前 top-k 的索引
            topk_tmp_val_buf[index] = total.u;      // 保存当前 top-k 的 log 概率值
            tmp_log_probs[total.p]  = -MAX_T_VAL;   // 将已经选过的值设为负无穷，以避免下次再选到
        }
        __syncthreads();
    }
}

// 执行 Top-k 采样的第二阶段。它从第一阶段选出的 Top-k 候选项中根据概率分布进行采样。
template<typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topk_stage2_sampling(const int* __restrict topk_tmp_id_buf,  // 第一阶段中的临时缓冲区，保存 Top-k 的候选索引
                                     T*             topk_tmp_val_buf,        // 第一阶段中的临时缓冲区，保存 Top-k 的候选概率值
                                     int*           ids,
                                     int*           sequence_length,
                                     bool*          finished,
                                     float*         cum_log_probs,
                                     float*         output_log_probs,
                                     const int      max_top_k,
                                     const int*     top_ks,                 // 每个批次的具体 Top-k 值，如果没有指定，使用 max_top_k
                                     const float    top_p,                  // 每个批次的具体 top_p 阈值
                                     const float*   top_ps,
                                     hiprandState_t* curandstate,            // CUDA 随机数生成器状态，用于生成随机数
                                     const int*     end_ids,
                                     const int      vocab_size,
                                     const bool*    skip_decode)
{
    const bool IS_FP16   = std::is_same<T, half>::value;
    const T    MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    const int tid      = threadIdx.x;
    const int batch_id = blockIdx.x;
    if (skip_decode != nullptr && skip_decode[batch_id]) {
        return;
    }

    const int   k              = (top_ks != nullptr) ? top_ks[batch_id] : max_top_k;
    const float prob_threshold = (top_ps != nullptr) ? top_ps[batch_id] : top_p;
    const int   size           = k * BLOCKS_PER_BEAM_;
    const int   stride         = max_top_k * BLOCKS_PER_BEAM_;
    // 初始化共享变量
    typedef hipcub::BlockReduce<TopK_2<float>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage         temp_storage;
    extern __shared__ char                               array[];
    __shared__ float                                     rand_num;
    __shared__ float                                     s_sum;
    __shared__ float                                     s_max;
    T*                                                   s_val = topk_tmp_val_buf + batch_id * stride;
    int*                                                 s_id  = reinterpret_cast<int*>(array);
    if (tid == 0) {
        s_sum = 0.0f;
    }
    TopK_2<float> partial;

    // 处理已经完成的序列：
    if (finished != nullptr && finished[batch_id] == true) {
        ids[batch_id] = end_ids[batch_id];
        return;
    }

    // 通过 softmax 处理 logits：
    float* s_val2 = reinterpret_cast<float*>(s_id + k);
    for (int ite = 0; ite < k; ite++) {
        partial.init();
#pragma unroll
        for (int i = tid; i < size; i += BLOCK_SIZE_) {
            partial.insert((float)s_val[i], i);
        }

        TopK_2<float> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op_2<float>);

        if (tid == 0) {
            if (ite == 0) {
                s_max = total.u;
            }
            s_id[ite]      = total.p;
            s_val[total.p] = -MAX_T_VAL;

            // when cum_log_probs are computed, topk_tmp_val_buf (logits_buf_) are already pre-processed by
            // softmax_kernel
            if (cum_log_probs == nullptr && output_log_probs == nullptr) {
                total.u = __expf(total.u - s_max);
            }
            s_val2[ite] = total.u;
            s_sum += total.u;
        }
        __syncthreads();
    }
    // 采样过程：
    if (tid == 0) {
        // 生成一个随机数 rand_num，用于在 softmax 归一化的概率分布中进行采样
        rand_num = (float)hiprand_uniform(curandstate + blockIdx.x) * prob_threshold * s_sum;
        // 从 Top-k 的候选项中依次减去 rand_num，直到 rand_num 变为负值或遍历完候选集，从而选择对应的候选项
        for (int i = 0; i < k; i++) {
            float exp_logit = s_val2[i];
            rand_num        = rand_num - exp_logit;
            if (rand_num <= 0.0f || i == k - 1) {
                // 如果 cum_log_probs 或 output_log_probs 被提供，函数会计算所选 token 的 log 概率，并更新累积 log 概率或输出 log 概率。
                // output_log_probs 保存的是所选 token 相对于整个 Top-k 集的概率，即 log P(i | i is in top-k)
                ids[batch_id] = topk_tmp_id_buf[batch_id * stride + s_id[i]] % vocab_size;
                if (cum_log_probs != nullptr || output_log_probs != nullptr) {
                    float log_prob = logf(exp_logit);
                    if (cum_log_probs != nullptr) {
                        cum_log_probs[batch_id] += log_prob;
                    }
                    if (output_log_probs != nullptr) {
                        // 'output_log_probs' is the probability induced by the top-k sampling.
                        // We normalize the probability 'exp_logit' of the selected token by
                        // the probability 's_sum' of a set of top-k tokens, meaning the log_prob
                        // is the probability of the selected token, conditioned on the event that
                        // it is selected, i.e.,
                        //   log_prob = log P(i | i is in top-k) = log(exp_logit / s_sum).
                        output_log_probs[batch_id] = log_prob - logf(s_sum);
                    }
                }
                break;
            }
        }
        // 更新序列状态
        if (sequence_length != nullptr && finished != nullptr) {
            sequence_length[batch_id] = finished[batch_id] ? sequence_length[batch_id] : sequence_length[batch_id] + 1;
            finished[batch_id]        = ids[batch_id] == end_ids[batch_id] ? true : false;
        }
    }
}




// CASE_K(1, 16, 128, 128, 8);
// CASE_K(17, 32, 256, 128, 8);
// topk_stage1 block内局部 topk ; topk_stage2_sampling 整体
#define CASE_K(K_MIN, K_MAX, BLOCK_SIZE_1_, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_)                                           \
    case K_MIN ... K_MAX:                                                                                              \
        topk_stage1<T, BLOCK_SIZE_1_, BLOCKS_PER_BEAM_>                                                                \
            <<<batch_size * BLOCKS_PER_BEAM_, BLOCK_SIZE_1_, 0, stream>>>(log_probs,                                   \
                                                                          temp_log_probs,                              \
                                                                          topk_tmp_id_buf,                             \
                                                                          topk_tmp_val_buf,                            \
                                                                          finished,                                    \
                                                                          max_top_k,                                   \
                                                                          top_ks,                                      \
                                                                          vocab_size,                                  \
                                                                          end_ids,                                     \
                                                                          skip_decode);                                \
        topk_stage2_sampling<T, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_>                                                       \
            <<<batch_size, BLOCK_SIZE_2_, K_MAX * sizeof(int) + K_MAX * sizeof(float), stream>>>(topk_tmp_id_buf,      \
                                                                                                 topk_tmp_val_buf,     \
                                                                                                 ids,                  \
                                                                                                 sequence_length,      \
                                                                                                 finished,             \
                                                                                                 cum_log_probs,        \
                                                                                                 output_log_probs,     \
                                                                                                 max_top_k,            \
                                                                                                 top_ks,               \
                                                                                                 top_p,                \
                                                                                                 top_ps,               \
                                                                                                 curandstate,          \
                                                                                                 end_ids,              \
                                                                                                 vocab_size,           \
                                                                                                 skip_decode);         \
        break;

template<typename T>
void invokeBatchTopKSampling(void*          workspace,
                             size_t&        workspace_size,
                             const T*       log_probs,
                             int*           ids,
                             int*           sequence_length,
                             bool*          finished,
                             float*         cum_log_probs,
                             float*         output_log_probs,
                             hiprandState_t* curandstate,
                             const int      max_top_k,
                             const int*     top_ks,
                             const float    top_p,
                             const float*   top_ps,
                             const int      vocab_size_padded,
                             const int*     end_ids,
                             hipStream_t   stream,
                             const int      batch_size,
                             const bool*    skip_decode)
{
    // Not allow an ambiguous inputs top_p and top_ps.
    assert(top_p == 1.0f || top_ps == nullptr);
    const int vocab_size              = vocab_size_padded;
    const int max_block_per_beam      = 8;
    int       temp_log_probs_buf_size = batch_size * vocab_size;                      // type float
    int       topk_tmp_ids_buf_size   = batch_size * max_top_k * max_block_per_beam;  // type int
    int       topk_tmp_val_buf_size   = batch_size * max_top_k * max_block_per_beam;  // type float

    // prevent memory misaligned address
    temp_log_probs_buf_size = (int)(ceil(temp_log_probs_buf_size / 4.)) * 4;
    topk_tmp_ids_buf_size   = (int)(ceil(topk_tmp_ids_buf_size / 4.)) * 4;
    topk_tmp_val_buf_size   = (int)(ceil(topk_tmp_val_buf_size / 4.)) * 4;

    if (workspace == nullptr) {
        workspace_size = sizeof(T) * temp_log_probs_buf_size + sizeof(int) * topk_tmp_ids_buf_size
                         + sizeof(T) * topk_tmp_val_buf_size;
        return;
    }

    T*   temp_log_probs   = (T*)workspace;
    int* topk_tmp_id_buf  = (int*)(temp_log_probs + temp_log_probs_buf_size);
    T*   topk_tmp_val_buf = (T*)(topk_tmp_id_buf + topk_tmp_ids_buf_size);

    switch (max_top_k) {
        CASE_K(1, 16, 128, 128, 8);    // 含义表示 k 1~16 时走这个分支
        CASE_K(17, 32, 256, 128, 8);
        CASE_K(33, 64, 256, 256, 8);
        CASE_K(65, 1024, 256, 256, 8);
        default:
            throw std::domain_error(fmtstr("top-k kernel supports 1<=k<=1024 but got k=%d", max_top_k));
    }
}

#undef CASE_K

template void invokeBatchTopKSampling(void*          workspace,
                                      size_t&        workspace_size,
                                      const float*   log_probs,
                                      int*           ids,
                                      int*           sequence_length,
                                      bool*          finished_buf,
                                      float*         cum_log_probs,
                                      float*         output_log_probs,
                                      hiprandState_t* curandstate,
                                      const int      max_top_k,
                                      const int*     top_ks,
                                      const float    top_p,
                                      const float*   top_ps,
                                      const int      vocab_size_padded,
                                      const int*     end_ids,
                                      hipStream_t   stream,
                                      const int      batch_size,
                                      const bool*    skip_decode);

template void invokeBatchTopKSampling(void*          workspace,
                                      size_t&        workspace_size,
                                      const half*    log_probs,
                                      int*           ids,
                                      int*           sequence_length,
                                      bool*          finished_buf,
                                      float*         cum_log_probs,
                                      float*         output_log_probs,
                                      hiprandState_t* curandstate,
                                      const int      max_top_k,
                                      const int*     top_ks,
                                      const float    top_p,
                                      const float*   top_ps,
                                      const int      vocab_size_padded,
                                      const int*     end_ids,
                                      hipStream_t   stream,
                                      const int      batch_size,
                                      const bool*    skip_decode);

template<typename T>
void invokeTopKSampling(void*          workspace,
                        size_t&        workspace_size,
                        const T*       log_probs,
                        int*           ids,
                        int*           sequence_length,
                        bool*          finished_buf,
                        float*         cum_log_probs,
                        float*         output_log_probs,
                        hiprandState_t* curandstate,
                        const int      top_k,
                        const float    top_p,
                        const int      vocab_size_padded,
                        const int*     end_ids,
                        hipStream_t   stream,
                        const int      batch_size,
                        const bool*    skip_decode)
{
    invokeBatchTopKSampling(workspace,
                            workspace_size,
                            log_probs,
                            ids,
                            sequence_length,
                            finished_buf,
                            cum_log_probs,
                            output_log_probs,
                            curandstate,
                            top_k,
                            nullptr,
                            top_p,
                            nullptr,
                            vocab_size_padded,
                            end_ids,
                            stream,
                            batch_size,
                            skip_decode);
}

template void invokeTopKSampling(void*          workspace,
                                 size_t&        workspace_size,
                                 const float*   log_probs,
                                 int*           ids,
                                 int*           sequence_length,
                                 bool*          finished_buf,
                                 float*         cum_log_probs,
                                 float*         output_log_probs,
                                 hiprandState_t* curandstate,
                                 const int      top_k,
                                 const float    top_p,
                                 const int      vocab_size_padded,
                                 const int*     end_ids,
                                 hipStream_t   stream,
                                 const int      batch_size,
                                 const bool*    skip_decode);

template void invokeTopKSampling(void*          workspace,
                                 size_t&        workspace_size,
                                 const half*    log_probs,
                                 int*           ids,
                                 int*           sequence_length,
                                 bool*          finished_buf,
                                 float*         cum_log_probs,
                                 float*         output_log_probs,
                                 hiprandState_t* curandstate,
                                 const int      top_k,
                                 const float    top_p,
                                 const int      vocab_size_padded,
                                 const int*     end_ids,
                                 hipStream_t   stream,
                                 const int      batch_size,
                                 const bool*    skip_decode);

template<typename T>
void invokeTopKTopPSampling(void*          workspace,
                            size_t&        workspace_size,
                            int*           output_ids,
                            const T*       logits,
                            int*           sequence_length,
                            bool*          finished_buf,
                            float*         cum_log_probs,
                            float*         output_log_probs,
                            hiprandState_t* curandstate,
                            const int      batch_size,
                            const int      top_k,
                            const float    top_p,
                            const int      vocab_size_padded,
                            const int*     end_ids,
                            hipStream_t   stream)
{
    // invokeTopKTopPSampling will be deprecated. Please use invokeTopKSampling instead.
    invokeTopKSampling(workspace,
                       workspace_size,
                       logits,
                       output_ids,
                       sequence_length,
                       finished_buf,
                       cum_log_probs,
                       output_log_probs,
                       curandstate,
                       top_k,
                       top_p,
                       vocab_size_padded,
                       end_ids,
                       stream,
                       batch_size,
                       nullptr);
}

template void invokeTopKTopPSampling(void*          workspace,
                                     size_t&        workspace_size,
                                     int*           output_ids,
                                     const float*   logits,
                                     int*           sequence_length,
                                     bool*          finished_buf,
                                     float*         cum_log_probs,
                                     float*         output_log_probs,
                                     hiprandState_t* curandstate,
                                     const int      batch_size,
                                     const int      top_k,
                                     const float    top_p,
                                     const int      vocab_size_padded,
                                     const int*     end_ids,
                                     hipStream_t   stream);

template void invokeTopKTopPSampling(void*          workspace,
                                     size_t&        workspace_size,
                                     int*           output_ids,
                                     const half*    logits,
                                     int*           sequence_length,
                                     bool*          finished_buf,
                                     float*         cum_log_probs,
                                     float*         output_log_probs,
                                     hiprandState_t* curandstate,
                                     const int      batch_size,
                                     const int      top_k,
                                     const float    top_p,
                                     const int      vocab_size_padded,
                                     const int*     end_ids,
                                     hipStream_t   stream);

}  // namespace fastertransformer
